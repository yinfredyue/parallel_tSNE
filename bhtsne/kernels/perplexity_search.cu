#include "hip/hip_runtime.h"
#include "perplexity_search.h"


__global__ void ComputePijKernel(
    volatile float *__restrict__ pij,
    const float *__restrict__ squared_dist,
    const float *__restrict__ betas,
    const unsigned int num_points,
    const unsigned int num_near_neighbors)
{
    register int TID, i, j;
    register float dist, beta;

    TID = threadIdx.x + blockIdx.x * blockDim.x;
    if (TID >= num_points * num_near_neighbors)
        return;

    i = TID / num_near_neighbors;
    j = TID % num_near_neighbors;

    beta = betas[i];
    dist = squared_dist[TID];

    // condition deals with evaluation of pii
    // FAISS neighbor zero is i so ignore it
    pij[TID] = (j == 0 & dist == 0.0f) ? 0.0f : __expf(-beta * dist); //TODO: This probably never evaluates to true
}


__global__ void PerplexitySearchKernel(
    volatile float *__restrict__ betas,
    volatile float *__restrict__ lower_bound,
    volatile float *__restrict__ upper_bound,
    volatile int *__restrict__ found,
    const float *__restrict__ neg_entropy,
    const float *__restrict__ row_sum,
    const float perplexity_target,
    const float epsilon,
    const int num_points)
{
    register int i, is_found;
    register float perplexity, neg_ent, sum_P, perplexity_diff, beta, min_beta, max_beta;
    i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= num_points)
        return;

    neg_ent = neg_entropy[i];
    sum_P = row_sum[i];
    beta = betas[i];

    min_beta = lower_bound[i];
    max_beta = upper_bound[i];

    perplexity = (neg_ent / sum_P) + __logf(sum_P);
    perplexity_diff = perplexity - __logf(perplexity_target);
    is_found = (perplexity_diff < epsilon && -perplexity_diff < epsilon);
    if (!is_found)
    {
        if (perplexity_diff > 0)
        {
            min_beta = beta;
            beta = (max_beta == FLT_MAX || max_beta == -FLT_MAX) ? beta * 2.0f : (beta + max_beta) / 2.0f;
        }
        else
        {
            max_beta = beta;
            beta = (min_beta == -FLT_MAX || min_beta == FLT_MAX) ? beta / 2.0f : (beta + min_beta) / 2.0f;
        }
        lower_bound[i] = min_beta;
        upper_bound[i] = max_beta;
        betas[i] = beta;
    }
    found[i] = is_found;
}


void SearchPerplexity(          hipblasHandle_t &handle,
                                thrust::device_vector<float> &pij,
                                thrust::device_vector<float> &squared_dist,
                                const float perplexity_target,
                                const float epsilon,
                                const int num_points,
                                const int num_near_neighbors)
{
    // use beta instead of sigma (this matches the bhtsne code but not the paper)
    // beta is just multiplicative instead of divisive (changes the way binary search works)
    printf("SearchPerplexity called\n");
    return;

    thrust::device_vector<float> betas(num_points, 1.0f);
    thrust::device_vector<float> lower_bound_beta(num_points, -FLT_MAX);
    thrust::device_vector<float> upper_bound_beta(num_points, FLT_MAX);
    thrust::device_vector<float> entropy(num_points * num_near_neighbors);
    thrust::device_vector<int> found(num_points);

    // TODO: this doesn't really fit with the style
    const int BLOCKSIZE1 = 1024;
    const int NBLOCKS1 = iDivUp(num_points * num_near_neighbors, BLOCKSIZE1);

    const int BLOCKSIZE2 = 128;
    const int NBLOCKS2 = iDivUp(num_points, BLOCKSIZE2);

    size_t iters = 0;
    int all_found = 0;
    thrust::device_vector<float> row_sum, neg_entropy;
    do
    {
        // compute Gaussian Kernel row
        ComputePijKernel<<<NBLOCKS1, BLOCKSIZE1>>>(
            thrust::raw_pointer_cast(pij.data()),
            thrust::raw_pointer_cast(squared_dist.data()),
            thrust::raw_pointer_cast(betas.data()),
            num_points, num_near_neighbors);
        GpuErrorCheck(hipDeviceSynchronize());

        // compute entropy of current row
        row_sum = tsnecuda::util::ReduceSum(handle, pij, num_near_neighbors, num_points, 0);
        thrust::transform(pij.begin(), pij.end(), entropy.begin(), tsnecuda::util::FunctionalEntropy());
        neg_entropy = tsnecuda::util::ReduceAlpha(handle, entropy, num_near_neighbors, num_points, -1.0f, 0);

        // binary search for beta
        PerplexitySearchKernel<<<NBLOCKS2, BLOCKSIZE2>>>(
            thrust::raw_pointer_cast(betas.data()),
            thrust::raw_pointer_cast(lower_bound_beta.data()),
            thrust::raw_pointer_cast(upper_bound_beta.data()),
            thrust::raw_pointer_cast(found.data()),
            thrust::raw_pointer_cast(neg_entropy.data()),
            thrust::raw_pointer_cast(row_sum.data()),
            perplexity_target, epsilon, num_points);
        GpuErrorCheck(hipDeviceSynchronize());

        // Check if searching is done
        all_found = thrust::reduce(found.begin(), found.end(), 1, thrust::minimum<int>());
        iters++;
    } while (!all_found && iters < 200);
    // TODO: Warn if iters == 200 because perplexity not found?

    tsnecuda::util::BroadcastMatrixVector(pij, row_sum, num_near_neighbors, num_points, thrust::divides<float>(), 1, 1.0f);
}


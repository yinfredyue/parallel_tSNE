/**
 * @brief Utilities for reduction across a matrix on an axis
 * 
 * @file reduce_utils.h
 * @author David Chan
 * @date 2018-04-04
 * Copyright (c) 2018, Regents of the University of California
 */

#ifndef SRC_INCLUDE_UTIL_REDUCE_UTILS_H_
#define SRC_INCLUDE_UTIL_REDUCE_UTILS_H_

#include "include/common.h"
#include "include/util/cuda_utils.h"

namespace tsnecuda {
namespace util {

/**
* @brief Reduce a matrix by summing then multiplying by alpha along the reduction axis
* 
* @param handle CUBLAS handle
* @param d_matrix The NxM matrix to reduce
* @param N The number of rows in the matrix
* @param M The number of columns in the matrix
* @param alpha The alpha to multiply by
* @param axis The axis to reduce on (0 = rows, 1 = cols)
* @return thrust::device_vector<float> The reduced vector 
*/
thrust::device_vector<float> ReduceAlpha(hipblasHandle_t &handle,
                            const thrust::device_vector<float> &d_matrix,
                            const int N,
                            const int M,
                            float alpha,
                            const int axis);

/**
* @brief Reduce a matrix by computing the mean of the reduction axis
* 
* @param handle CUBLAS handle
* @param d_matrix The NxM matrix to reduce
* @param N The number of rows in the matrix
* @param M The number of columns in the matrix
* @param axis The axis to reduce on (0 = rows, 1 = cols)
* @return thrust::device_vector<float> The reduced vector 
*/
thrust::device_vector<float> ReduceMean(hipblasHandle_t &handle,
                                const thrust::device_vector<float> &d_matrix,
                                const int N,
                                const int M,
                                const int axis);

/**
* @brief Reduce a matrix by computing the sum of the reduction axis
* 
* @param handle CUBLAS handle
* @param d_matrix The NxM matrix to reduce
* @param N The number of rows in the matrix
* @param M The number of columns in the matrix
* @param axis The axis to reduce on (0 = rows, 1 = cols)
* @return thrust::device_vector<float> The reduced vector 
*/
thrust::device_vector<float> ReduceSum(hipblasHandle_t &handle,
                                const thrust::device_vector<float> &d_matrix,
                                const int N,
                                const int M,
                                const int axis);

}  // namespace util
}  // namespace tsnecuda

#endif  // SRC_INCLUDE_UTIL_REDUCE_UTILS_H_

